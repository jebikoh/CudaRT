#include "hip/hip_runtime.h"
#include <iostream>
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include "sphere.hpp"
#include "rt.hpp"
#include "hittable.hpp"
#include "hittablelist.hpp"
#include "stb_image_write.h"
#include "camera.hpp"


using Color = jtx::Vec3f;
using RGB8 = jtx::Vec3<uint8_t>;

void savePNG(const char *path, const RGB8 *fb, int width, int height) {}


#define CHECK_CUDA(val) check_cuda( (val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}


__device__ jtx::Vec3f rayColor(const jtx::Rayf &r, Hittable **world) {
    HitRecord rec;
    if ((*world)->hit(r, {0.0f, jtx::INFINITY_F}, rec)) {
        return 0.5f * (rec.normal + 1.0f);
    }
    float t = 0.5f * (jtx::normalize(r.dir).y + 1.0f);
    return jtx::lerp(jtx::Vec3f{1.0f, 1.0f, 1.0f}, jtx::Vec3f{0.5f, 0.7f, 1.0f}, t);
}

__global__ void createWorld(Hittable **d_list, Hittable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new Sphere(jtx::Point3f{0, 0, -1}, 0.5);
        d_list[1] = new Sphere(jtx::Point3f{0, -100.5, -1}, 100);
        *d_world = new HittableList(d_list, 2);
    }
}

__global__ void freeWorld(Hittable **d_list, Hittable **d_world) {
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= maxX) || (j >= maxY)) return;
    int pixelIndex = j * maxX + i;
    hiprand_init(1984, pixelIndex, 0, &randState[pixelIndex]);
}

__global__ void render(RGB8 *fb,
                       int maxX,
                       int maxY,
                       int samplesPerPixel,
                       jtx::Vec3f lowerLeft,
                       jtx::Vec3f horizontal,
                       jtx::Vec3f vertical,
                       jtx::Vec3f origin,
                       Hittable **world,
                       hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= maxX) || (j >= maxY)) return;
    int pixel_index = j * maxX + i;
    hiprandState localRandState = randState[pixel_index];
    Color color(0, 0, 0);
    for (int s = 0; s < samplesPerPixel; ++s) {
        float u = (float(i) + hiprand_uniform(&localRandState)) / float(maxX);
        float v = (float(j) + hiprand_uniform(&localRandState)) / float(maxY);
        jtx::Rayf r(origin, lowerLeft + u * horizontal + v * vertical);
        color += rayColor(r, world);
    }
    color /= float(samplesPerPixel);
    fb[pixel_index].x = uint8_t(256 * jtx::clamp(color.r, 0.0f, 0.999f));
    fb[pixel_index].y = uint8_t(256 * jtx::clamp(color.g, 0.0f, 0.999f));
    fb[pixel_index].z = uint8_t(256 * jtx::clamp(color.b, 0.0f, 0.999f));
}

int main() {
    const int nx = 1200;
    const int ny = 600;
    const int tx = 8;
    const int ty = 8;
    const int samplesPerPixel = 50;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int numPixels = nx * ny;
    auto fb_size = numPixels * sizeof(RGB8);

    RGB8 *fb;
    CHECK_CUDA(hipMallocManaged((void **) &fb, fb_size));

    Hittable **d_list;
    CHECK_CUDA(hipMalloc((void **) &d_list, 2 * sizeof(Hittable *)));
    Hittable **d_world;
    CHECK_CUDA(hipMalloc((void **) &d_world, sizeof(Hittable *)));
    hiprandState *d_randState;
    CHECK_CUDA(hipMalloc((void **) &d_randState, numPixels * sizeof(hiprandState)));

    createWorld<<<1, 1>>>(d_list, d_world);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    renderInit<<<blocks, threads>>>(nx, ny, d_randState);

    clock_t start, stop;
    start = clock();
    render<<<blocks, threads>>>(fb, nx, ny, samplesPerPixel,
                                jtx::Vec3f(-2.0, -1.0, -1.0),
                                jtx::Vec3f(4.0, 0.0, 0.0),
                                jtx::Vec3f(0.0, 2.0, 0.0),
                                jtx::Vec3f(0.0, 0.0, 0.0),
                                d_world,
                                d_randState);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double) (stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Time: " << timer_seconds << " seconds\n";

    stbi__flip_vertically_on_write = 1;
    stbi_write_png("output.png", nx, ny, 3, fb, nx * 3);

    CHECK_CUDA(hipDeviceSynchronize());
    freeWorld<<<1,1>>>(d_list,d_world);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipFree(d_list));
    CHECK_CUDA(hipFree(d_world));
    CHECK_CUDA(hipFree(fb));
}
